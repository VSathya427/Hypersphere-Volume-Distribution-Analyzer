#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NUM_INTERVALS 100
#define NUM_DIMENSIONS 16
#define NUM_SAMPLES 30000
#define BLOCK_SIZE 256

// Function to generate a random number between 0 and 1
__host__ __device__ double rand_double(unsigned int &seed)
{
    seed = (seed * 1103515245 + 12345) & 0x7fffffff;
    return (double)seed / (double)0x7fffffff;
}

// Function to compute the squared Euclidean norm of a vector
__device__ double squared_norm(double *v, int dim)
{
    double sum = 0.0;
    for (int i = 0; i < dim; i++)
    {
        sum += v[i] * v[i];
    }
    return sum;
}

// Kernel function to generate random points and compute histogram
__global__ void ball_samp_kernel(int *histogram, int dim, int num_samples, unsigned int *seeds)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_samples)
        return;

    double point[NUM_DIMENSIONS];
    double norm_squared;
    unsigned int seed = seeds[idx];

    do
    {
        for (int j = 0; j < dim; j++)
        {
            point[j] = 2.0 * rand_double(seed) - 1.0;
        }
        norm_squared = squared_norm(point, dim);
    } while (norm_squared > 1.0);

    double distance = sqrt(1.0 - norm_squared);
    int bin = (int)(distance * NUM_INTERVALS);
    atomicAdd(&histogram[bin], 1);
}

// Sequential implementation
void ball_samp_cpu()
{
    for (int dim = 2; dim <= NUM_DIMENSIONS; dim++)
    {
        printf("Dimension: %d\n", dim);
        int histogram[NUM_INTERVALS] = {0};

        unsigned int *seeds;
        hipMallocManaged(&seeds, NUM_SAMPLES * sizeof(unsigned int));
        for (int i = 0; i < NUM_SAMPLES; i++)
        {
            seeds[i] = rand();
        }

        int *d_histogram;
        hipMalloc(&d_histogram, NUM_INTERVALS * sizeof(int));
        hipMemset(d_histogram, 0, NUM_INTERVALS * sizeof(int));

        int grid_size = (NUM_SAMPLES + BLOCK_SIZE - 1) / BLOCK_SIZE;
        ball_samp_kernel<<<grid_size, BLOCK_SIZE>>>(d_histogram, dim, NUM_SAMPLES, seeds);
        hipDeviceSynchronize();

        hipMemcpy(histogram, d_histogram, NUM_INTERVALS * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < NUM_INTERVALS; i++)
        {
            double fraction = (double)histogram[i] / (double)NUM_SAMPLES;
            printf("%lf ", fraction);
        }
        printf("\n");

        hipFree(d_histogram);
        hipFree(seeds);
    }
}

int main(int argc, char *argv[])
{
    srand(time(NULL));
    ball_samp_cpu();
    return 0;
}